#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "jr_common.h" // ...

#ifdef GPUDEBUG
    #define debug_printf(...) printf(__VA_ARGS__)
#else
    #define debug_printf(...)
#endif

	// Helper /////////////////////////////////////////////////////////////////////
	// Checking return types of all CUDA runtime functions is best practice, 
	//  ... has negligible performance impact and should not be omitted unless absolutely necessary
	__host__ inline
	void __cudaSafeCall(hipError_t err, const char *file, const int line, char const *call=nullptr) { // Actual check function
		if (hipSuccess != err) {
			fprintf(stderr, "[ERROR] CUDA call%s%s at %s:%d\n%s\n", call?" to ":"", call, file, line, hipGetErrorString(err));
			exit(0);
		}
	} // __cudaSafeCall
    #define cuCheck(err) __cudaSafeCall((err), __FILE__, __LINE__, #err) // Syntactic sugar to enhance output

    // As CUDA kernel launches are asynchronous error checking is more difficult, 
    // ... as the check might occur prior to the actual error - this macro makes 
    // ... sure it catches an error if it occurs by explicit Synchronization. 
    // ... Due to the performance impact it is only active in debug mode.
    __host__ inline
    void __cuKernelCheck(const char* file, const int line) {
#ifdef GPUDEBUG
		hipDeviceSynchronize();
		hipError_t err = hipPeekAtLastError();
		if (hipSuccess != err) {
			fprintf(stderr, "[ERROR] CUDA kernel call at %s:%d\n%s\n",  file, line, hipGetErrorString(err));
			exit(0);
		} // err
#endif
	} // __cuKernelCheck
    #define cuKernelCheck() __cuKernelCheck(__FILE__, __LINE__)

	// GPU Memory management /////////////////////////////////////////////////////////

    __host__
    void copy_data_to_GPU(void *d, void const *h, size_t const nBytes, hipStream_t const stream) {
        debug_printf("[INFO] transfer %lu Byte from %p @host to %p @device\n", nBytes, h, d);
        cuCheck(hipMemcpyAsync(d, h, nBytes, hipMemcpyHostToDevice, stream));
    } // copy_data_to_GPU

    __host__
    void get_data_from_GPU(void *h, void const *d, size_t const nBytes, hipStream_t const stream) {
        debug_printf("[INFO] transfer %lu Byte from %p @device to %p @host\n", nBytes, d, h);
        cuCheck(hipMemcpyAsync(h, d, nBytes, hipMemcpyDeviceToHost, stream));
    } // get_data_from_GPU

    __host__
    void* __allocate_on_GPU(size_t const nBytes, char const *srcfile=nullptr, int const srcline=0) {
        debug_printf("[INFO] hipMalloc %.6f MByte in %s:%i\n", 1e-6*nBytes, srcfile, srcline);
        void* d = nullptr;
        cuCheck(hipMalloc(&d, nBytes));
        return d;
    } // allocate_on_GPU
    #define malloc_GPU(TYPE, NUM) (TYPE *)__allocate_on_GPU((NUM)*sizeof(TYPE), __FILE__, __LINE__)

    __host__
    void free_memory_on_GPU(void**d) {
        cuCheck(hipFree(*d));
        *d = nullptr;
    } // free_memory_on_GPU

    
    __host__
    void* __allocate_unified_memory(size_t const nBytes, char const *srcfile=nullptr, int const srcline=0) {
        debug_printf("[INFO] hipMallocManaged %.6f MByte in %s:%i\n", 1e-6*nBytes, srcfile, srcline);
        void* d = nullptr;
        cuCheck(hipMallocManaged(&d, nBytes));
        return d;
    } // allocate_on_GPU
    #define getUnifiedMemory(TYPE, NUM) (TYPE *)__allocate_unified_memory((NUM)*sizeof(TYPE), __FILE__, __LINE__)
    
    __host__
	tbl_t* get_tbl_on_GPU(ctl_t const *ctl) {
		static tbl_t *tbl_G = nullptr;
		if (!tbl_G) {
			tbl_t* tbl = get_tbl(ctl);
#ifdef  USE_UNIFIED_MEMORY_FOR_TABLES
            printf("[INFO] allocated %.3f MByte unified memory for tables\n", 1e-6*sizeof(tbl_t));
            tbl_G = tbl; // just passing a pointer, same memory space
#else
            printf("[INFO] try to allocate %.3f MByte GPU memory for tables\n", 1e-6*sizeof(tbl_t));
			tbl_G = malloc_GPU(tbl_t, 1);
			copy_data_to_GPU(tbl_G, tbl, sizeof(tbl_t), 0);
#endif
		} // !tbl_G
		return tbl_G;
	} // get_tbl_on_GPU

	// ################ GPU driver routines - keep consistent with CPUdrivers.cu ##############

	// Radiance -> Brightness conversion //////////////////////////////////////////
	void __global__ // GPU-kernel
		radiance_to_brightness_GPU(ctl_t const *ctl, obs_t *obs) { // operates onto obs in-place
			for(int ir = blockIdx.x; ir < obs->nr; ir += gridDim.x) { // grid stride loop over blocks = rays
				for(int id = threadIdx.x; id < ctl->nd; id += blockDim.x) { // grid stride loop over threads = detectors
                    auto const radiance = obs->rad[ir][id];
					obs->rad[ir][id] = brightness_core(radiance, ctl->nu[id]); // modify in-place
				} // id
			} // ir
		} // radiance_to_brightness_GPU

	// Add planetary surface emission ////////////////////////////////////////////
	void __global__ // GPU-kernel
		surface_terms_GPU(const tbl_t *tbl, obs_t *obs, double const tsurf[], int const nd) {
			for(int ir = blockIdx.x; ir < obs->nr; ir += gridDim.x) { // grid stride loop over blocks = rays
				for(int id = threadIdx.x; id < nd; id += blockDim.x) { // grid stride loop over threads = detectors
					add_surface_core(obs, tbl, tsurf[ir], ir, id);
				} // id
			} // ir
		} // surface_terms_GPU

// template<int CO2, int H2O, int N2, int O2> for multi-versioning
#define KERNEL "jr_fusion_kernel.mv4g.cu"
      #include "jr_multiversion4gases.h" // fusion_kernel_GPU_0000, _0001, ..., _1111
#undef  KERNEL

    __host__
	void multi_version_GPU(char const fourbit, tbl_t const *tbl, ctl_t const *ctl,
			obs_t *obs, pos_t const (*restrict los)[NLOS],
			int const np[], int const ig_co2, int const ig_h2o,
			unsigned const grid, unsigned const block, unsigned const shmem, hipStream_t const stream) {
#define LaunchKernel <<< grid, block, shmem, stream >>> (tbl, ctl, obs, los, np, ig_co2, ig_h2o)
		switch (fourbit) {
			case 0b0000: fusion_kernel_GPU_0000 LaunchKernel; break;
			case 0b0001: fusion_kernel_GPU_0001 LaunchKernel; break;
			case 0b0010: fusion_kernel_GPU_0010 LaunchKernel; break;
			case 0b0011: fusion_kernel_GPU_0011 LaunchKernel; break;
			case 0b0100: fusion_kernel_GPU_0100 LaunchKernel; break;
			case 0b0101: fusion_kernel_GPU_0101 LaunchKernel; break;
			case 0b0110: fusion_kernel_GPU_0110 LaunchKernel; break;
			case 0b0111: fusion_kernel_GPU_0111 LaunchKernel; break;
			case 0b1000: fusion_kernel_GPU_1000 LaunchKernel; break;
			case 0b1001: fusion_kernel_GPU_1001 LaunchKernel; break;
			case 0b1010: fusion_kernel_GPU_1010 LaunchKernel; break;
			case 0b1011: fusion_kernel_GPU_1011 LaunchKernel; break;
			case 0b1100: fusion_kernel_GPU_1100 LaunchKernel; break;
			case 0b1101: fusion_kernel_GPU_1101 LaunchKernel; break;
			case 0b1110: fusion_kernel_GPU_1110 LaunchKernel; break;
			case 0b1111: fusion_kernel_GPU_1111 LaunchKernel; break;
		} // fourbit
#undef	LaunchKernel
	} // multi_version_GPU

	// Raytracing ////////////////////////////////////////////////////////////////
	void __global__ // GPU-kernel
		raytrace_rays_GPU(ctl_t const *ctl, const atm_t *atm, obs_t *obs, pos_t los[][NLOS], double *tsurf, int np[]) {
			for(int ir = blockIdx.x*blockDim.x + threadIdx.x; ir < obs->nr; ir += blockDim.x*gridDim.x) { // grid stride loop over rays
				np[ir] = traceray(ctl, &atm[0], obs, ir, los[ir], &(tsurf[ir]));
			} // ir
		} // raytrace_rays_GPU

	// Compute hydrostatic equilibria for all atm //////////////////////////////
	void __global__ // GPU-kernel
		hydrostatic_kernel_GPU(ctl_t const *ctl, atm_t *atm, const int nr, int const ig_h2o) {
			for(int ir = blockIdx.x*blockDim.x + threadIdx.x; ir < nr; ir += blockDim.x*gridDim.x) {
				hydrostatic_1d_h2o(ctl, &atm[0], 0, atm[0].np, ig_h2o);
			} // ip
		} // hydrostatic_kernel

    __host__
	void hydrostatic1d_GPU(ctl_t const *ctl, ctl_t const *ctl_G,
			atm_t *atm_G, int const nr, int const ig_h2o, hipStream_t const stream) {
		if(ctl->hydz < 0) return; // Check reference height
		hydrostatic_kernel_GPU<<<nr/32 + 1, 32, 0, stream>>> (ctl_G, atm_G, nr, ig_h2o);
	} // hydrostatic1d_GPU

	// ################ end of GPU driver routines ##############

	// GPU control struct containing GPU version of input, intermediate and output arrays
	typedef struct {
		obs_t  *obs_G;
		atm_t  *atm_G;
		pos_t (*los_G)[NLOS];
		double *tsurf_G;
		int    *np_G;
		hipStream_t stream;
	} gpuLane_t;

	// The full forward model working on one package of NR rays
    __host__
	void formod_one_package(ctl_t const *ctl, ctl_t const *ctl_G,
			tbl_t const *tbl_G,
			atm_t const *atm, // can be made const if we do not get the atms back
			obs_t *obs,
			gpuLane_t const *gpu)
    // a workload manager for the GPU
    {
		debug_printf("[INFO] %s GPU\n"
               " Rays:    %9d (max %d)\n"
               " Gases:   %9d (max %d)\n"
               " Channels:%9d (max %d)\n",
               __func__, obs->nr, NR, ctl->ng, NG, ctl->nd, ND);
        
		atm_t *atm_G = gpu->atm_G;
		obs_t *obs_G = gpu->obs_G;
		pos_t (* los_G)[NLOS] = gpu->los_G;
		double *tsurf_G = gpu->tsurf_G;
		int *np_G = gpu->np_G;
		hipEvent_t finishedEvent;
		hipEventCreate(&finishedEvent);

		// gas absorption continua configuration
		static int ig_co2 = -999, ig_h2o = -999;
		if((ctl->ctm_h2o) && (ig_h2o == -999)) ig_h2o = find_emitter(ctl, "H2O");
		if((ctl->ctm_co2) && (ig_co2 == -999)) ig_co2 = find_emitter(ctl, "CO2");
		// binary switches for the four gases
		char const fourbit = (char)
                ( ( (1 == ctl->ctm_co2) && (ig_co2 >= 0) )*0b1000   // CO2
                + ( (1 == ctl->ctm_h2o) && (ig_h2o >= 0) )*0b0100   // H2O
                +   (1 == ctl->ctm_n2)                    *0b0010   // N2
                +   (1 == ctl->ctm_o2)                    *0b0001); // O2

		unsigned const nd = ctl->nd, nr = obs->nr; // abbreviate

		hipStream_t stream = gpu->stream;
		copy_data_to_GPU(atm_G, atm, 1*sizeof(atm_t), stream);
		copy_data_to_GPU(obs_G, obs, 1*sizeof(obs_t), stream);
        
        
        for(int benchmark = 0; benchmark < 100; ++benchmark) {
        
		hydrostatic1d_GPU(ctl, ctl_G, atm_G, nr, ig_h2o, stream); // in this call atm_G gets modified
		cuKernelCheck();
		raytrace_rays_GPU <<< (nr/64)+1, 64, 0, stream>>> (ctl_G, atm_G, obs_G, los_G, tsurf_G, np_G);
		cuKernelCheck();
		multi_version_GPU(fourbit, tbl_G, ctl_G, obs_G, los_G, np_G, ig_co2, ig_h2o, nr, nd, ctl->gpu_nbytes_shared_memory, stream);
		cuKernelCheck();
		surface_terms_GPU <<< nr, nd, 0, stream>>> (tbl_G, obs_G, tsurf_G, nd);
		cuKernelCheck();
        
        } // benchmark
        
        if (ctl->write_bbt) { // convert radiance to brightness (in-place)
            radiance_to_brightness_GPU <<< nr, nd, 0, stream >>> (ctl_G, obs_G);
        } // write_bbt

// 		get_data_from_GPU(atm, atm_G, 1*sizeof(atm_t), stream); // do we really need to get the atms back?
		get_data_from_GPU(obs, obs_G, 1*sizeof(obs_t), stream); // always transfer NR rays

		// Wait for GPU operations to complete
		cuCheck(hipEventRecord(finishedEvent, stream));
		cuCheck(hipEventSynchronize(finishedEvent));

	} // formod_one_package

    // make sure that formod_GPU can be linked from CPUdrivers.c
	extern "C" {
      void formod_GPU(ctl_t const *ctl, atm_t *atm, obs_t *obs);
    }

	extern "C" {
      int omp_get_thread_num();
    }
    
	__host__
	void formod_GPU(ctl_t const *ctl, atm_t *atm, obs_t *obs) {
		static ctl_t *ctl_G=NULL;
		static tbl_t *tbl_G=NULL;

		static int numDevices = 0;
		static gpuLane_t* gpuLanes=NULL;
		static size_t numLanes = 0;
		static size_t nextLane = 0;
		size_t myLane = 0;

		static bool do_init = true;
		bool early_return = false;

#pragma omp critical
		{
			if (do_init) {
				size_t const sizePerLane = sizeof(obs_t) + NR * (sizeof(atm_t) + sizeof(pos_t[NLOS]) + sizeof(double) + sizeof(int));
              
              if (ctl->checkmode) {
                printf("# %s: GPU memory requirement per lane is %.3f MByte\n", __func__, 1e-6*sizePerLane);
              } else {
				cuCheck(hipGetDeviceCount(&numDevices));
				if(ctl->MPIlocalrank > numDevices) {
					fprintf(stderr, "More MPI-Ranks on Node than GPUs. Abort.\n");
					exit(1);
				}
				cuCheck(hipSetDevice(ctl->MPIlocalrank));

				// Initialize ctl and tbl-struct (1 per GPU)
				ctl_G = malloc_GPU(ctl_t, 1);
				copy_data_to_GPU(ctl_G, ctl, sizeof(ctl_t), 0);

				tbl_G = get_tbl_on_GPU(ctl);

				// Get number of possible lanes
				size_t gpuMemFree, gpuMemTotal;
				cuCheck(hipMemGetInfo(&gpuMemFree, &gpuMemTotal));
                debug_printf("[INFO] memory GPU: free %.3f of total %.3f MByte = %.1f %%\n",
                      1e-6*gpuMemFree, 1e-6*gpuMemTotal, gpuMemFree/(.01*gpuMemTotal));
              
				numLanes = (size_t)((0.9*gpuMemFree) / (double)sizePerLane); // Only use 90% of free GPU memory ...
                                                  // ... other space is needed for alignment and profiling buffers
				size_t const maxNumLanes = 4; // Do not really need more than a handfull of lanes
				if (numLanes > maxNumLanes) numLanes = maxNumLanes;
                debug_printf("[INFO] GPU memory per lane: %.3f MByte, try to fit %i lanes\n", 1e-6*sizePerLane, numLanes);
				if (numLanes < 1) ERRMSG("Memory requirement per lane is too high, no lanes");

				gpuLanes = (gpuLane_t*) malloc(numLanes*sizeof(gpuLane_t)); // (this memory is never freed)
				for(size_t lane = 0; lane < numLanes; ++lane) {
					gpuLane_t* gpu = &(gpuLanes[lane]); // abbreviation
					// Allocation of GPU memory
					gpu->obs_G		= malloc_GPU(obs_t, 1);
					gpu->atm_G		= malloc_GPU(atm_t, NR);
					gpu->tsurf_G	= malloc_GPU(double, NR);
					gpu->np_G		= malloc_GPU(int, NR);
					gpu->los_G		= (pos_t (*)[NLOS])__allocate_on_GPU(NR*NLOS*sizeof(pos_t), __FILE__, __LINE__); 
                                      // for los_G[NLOS], the macro malloc_GPU does not work
					cuCheck(hipStreamCreate(&gpu->stream));
                    debug_printf("[INFO] hipStreamCreate --> streamId %d\n", gpu->stream);
				} // lane
              } // checkmode

				do_init = false;
				nextLane = 0;
#ifdef RETURN_AFTER_INIT
				early_return = true;
#endif 				
			} // do_init

			// Save own Lane and increment global / static counter
			myLane = nextLane;
			nextLane++;
			if(nextLane >= numLanes) nextLane=0;
		} // omp critical

		if (ctl->checkmode) { printf("# %s: no operation in checkmode\n", __func__); return; }
		
		if (early_return) {
			printf("# %s: no operation after initialization (benchmarking mode)\n", __func__);
			return;
		} // early_return

		cuCheck(hipSetDevice(ctl->MPIlocalrank));

		char mask[NR][ND];
		save_mask(mask, obs, ctl);
#pragma omp parallel
        {
            int const cpu_thread_id = omp_get_thread_num();
#pragma omp parallel for num_threads(numDevices)
            for(int gpu_id = 0; gpu_id < numDevices; ++gpu_id) {
                debug_printf("# gpu_id=%i runs one package started by CPU thread %i\n", gpu_id, cpu_thread_id);	
                cuCheck(hipSetDevice(gpu_id));
                copy_data_to_GPU(ctl_G, ctl, sizeof(ctl_t), gpuLanes[myLane].stream); // controls might change, update
                formod_one_package(ctl, ctl_G, tbl_G, atm, obs, &gpuLanes[myLane]);
            } // gpu_id
        }
		apply_mask(mask, obs, ctl);
	} // formod_GPU
